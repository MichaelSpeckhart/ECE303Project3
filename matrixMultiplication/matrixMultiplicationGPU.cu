#include "hip/hip_runtime.h"
/*
Luke Hale and Michael Speckhart
Project #3
Part #2.7
3/8/2024
*/
#include "timer.h"
#include <iostream>
#include <iomanip>
#include <random>

using namespace std;

const size_t MATRIX_SIZE = 100;

__global__ void mult_matrix_kernel(double* m1Buf,double* m2Buf,double* m3Buf)
{
    double (*m1)[MATRIX_SIZE] = (double (*)[MATRIX_SIZE])m1Buf;
    double (*m2)[MATRIX_SIZE] = (double (*)[MATRIX_SIZE])m2Buf;
    double (*m3)[MATRIX_SIZE] = (double (*)[MATRIX_SIZE])m3Buf;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < MATRIX_SIZE; i += stride)
        for (size_t j = 0; j < MATRIX_SIZE; ++j){
            for (size_t k = 0; k < MATRIX_SIZE; ++k)
                m3[i][j] += m1[i][k] * m2[k][j];
        }
}

int main(){

    size_t bytes = MATRIX_SIZE * MATRIX_SIZE * sizeof(double);
    //random num generator
    random_device rd;
    default_random_engine eng(rd());
    uniform_real_distribution<double> distr(-10000, 10000);
    //malloc m1
    double* m1Buf = (double*)malloc(bytes);
    double (*m1)[MATRIX_SIZE] = (double (*)[MATRIX_SIZE])m1Buf;
    //fill m1
    for(size_t i = 0 ; i < MATRIX_SIZE;i++){
        for(size_t j = 0 ; j <MATRIX_SIZE; j++){
            m1[i][j] = distr(eng);
        }
    }

    //malloc m2
    double* m2Buf = (double*)malloc(bytes);
    double (*m2)[MATRIX_SIZE] = (double (*)[MATRIX_SIZE])m2Buf;
    //fill m1
    for(size_t i = 0 ; i < MATRIX_SIZE;i++){
        for(size_t j = 0 ; j <MATRIX_SIZE; j++){
            m2[i][j] = distr(eng);
        }
    }

    //malloc m3
    double* m3BufDevice;
    hipMalloc(&m3BufDevice, bytes);

    double* m1BufDevice;
    hipMalloc(&m1BufDevice, bytes);
    hipMemcpy(m1BufDevice, m1Buf, bytes, hipMemcpyHostToDevice);

    double* m2BufDevice;
    hipMalloc(&m2BufDevice, bytes);
    hipMemcpy(m2BufDevice, m2Buf, bytes, hipMemcpyHostToDevice);

    //double (*m3)[MATRIX_SIZE] = (double (*)[MATRIX_SIZE])m3Buf;
    int threads_per_block = 512;
    int deviceId;
    hipGetDevice(&deviceId);
  
    int numberOfSMs;
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    
    int number_of_blocks = 32 * numberOfSMs;

    StartTimer();
    mult_matrix_kernel<<<number_of_blocks, threads_per_block>>>(m1BufDevice, m2BufDevice, m3BufDevice);
    std::cout << GetTimer() << " ms for multiplication of " <<MATRIX_SIZE <<" size"<< std::endl;\

    double* m3Buf = (double*)malloc(bytes);
    hipMemcpy(m3Buf, m3BufDevice, bytes, hipMemcpyDeviceToHost);
    //double (*m3)[MATRIX_SIZE] = (double (*)[MATRIX_SIZE])m3Buf;

    free(m1Buf);
    free(m2Buf);
    free(m3Buf);
    hipFree(m1BufDevice);
    hipFree(m2BufDevice);
    hipFree(m3BufDevice);
}
